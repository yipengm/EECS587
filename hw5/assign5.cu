
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <algorithm>
#include <stdio.h>
#include<float.h>


#define THREADS_PER_BLOCK 1024           //max of the threads in one block is 1024

// Kernel function to add the elements of two arrays
__global__ 
void iteration(double *d_A,double *d_B,int n)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x; 
    
    if(i<n*n){
      if((0<i&&i<n)||i%n==0||(i+1)%n==0||(n*n-n<i&&i<n*n-1)){
        d_B[i]=d_A[i];
      }
      else{
        double local[4];
        double temp;
        double first_small;
        double secnd_small;


        local[0]=d_A[i+n-1];
        local[1]=d_A[i+n+1];
        local[2]=d_A[i-n-1];
        local[3]=d_A[i-n+1];

        if (local[0]>local[1]){
          first_small = local[1];
          secnd_small = local[0];
        }
        else{
          first_small = local[0];
          secnd_small = local[1];
        }

        if(local[2]<first_small){
          secnd_small = first_small;
          first_small = local[2];
        }
        else if(local[2]<secnd_small){
          secnd_small = local[2];
        }

        if(local[3]<first_small){
          secnd_small = first_small;
          first_small = local[3];
        }
        else if(local[3]<secnd_small){
          secnd_small = local[3];
        }
        d_B[i]=secnd_small+d_A[i];
      }
    }    
}

__global__ 
void sumblock(double *d_A, int size, double*sum_temp)
{

  extern __shared__ double sum_block[];

  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int k;
  int bound;

  double temp=0;
  if(i<size){
    temp=d_A[i]+temp;
  }

  sum_block[threadIdx.x]=temp;
  __syncthreads();

  k = blockDim.x;
  while(k>1){
    if(k%2==0){
      bound=k/2;
      if (threadIdx.x<bound){
        sum_block[threadIdx.x]=sum_block[threadIdx.x]+sum_block[threadIdx.x+bound];
      }
      __syncthreads();
      k=k/2;
    }
    else{
      bound=k/2;
      if (threadIdx.x<=bound||threadIdx.x!=0){
        sum_block[threadIdx.x]=sum_block[threadIdx.x]+sum_block[threadIdx.x+bound];
      }
      __syncthreads();
      k=k/2+1;
    }             
  } 

  if(threadIdx.x==0){
    sum_temp[blockIdx.x] =sum_block[0];
  }
}

int main(int argc, char **argv)
{
  int n = atoi(argv[1]);          //The size of the matrix
  
  double *A;                      //The definition of matrix A
  double *d_A;                    //The definition of matrix A in gpu
  double *d_B;                    //The definition of iterated matrix A in gpu

  hipEvent_t start;              // the start time of gpu calculation
  hipEvent_t end;                // the end time of gpu calculation
  float elapsedTime=0;            // the elapsed time of gpu calculation

  double sum=0;                   //The sum of the matrix A after 10 iterations
  double *sum_temp;               //The sum of the matrix A after 10 iterations for each block on gpu
  double center=0;                //The center of the matrix A after 10 iterations
  double verification=0;          //The A(17,31) of the matrix A after 10 iterations

  int count=0;                    //To count the number of the iterations

  int grid;                       //The dimension of the grid on the gpu
  int block;                      //The dimension of the block on the gpu
  
  int size;                       //The size of elements in each iteration for sum
  double* sum_ptr;                //The pointer of the first element in each iteration for sum

  
  hipEventCreate(&start);  
  hipEventCreate(&end);

  // Allocate Unified Memory on the CPU
  A = (double*)malloc(n*n*sizeof(double));
  
  //initialize the matrix A on the host
  for(int i=0;i<n;i++){
    for(int j=0;j<n;j++){
      A[i*n+j]=(1+cos(2*i)+sin(j))*(1+cos(2*i)+sin(j));
    }
  }

  // Allocate Memory on the GPU
  hipMalloc(&d_A, n*n*sizeof(double)); 
  hipMalloc(&d_B, n*n*sizeof(double));  

  //copy the data to the gpu from cpu
  hipMemcpy(d_A, A, n*n*sizeof(double), hipMemcpyHostToDevice);

  block=THREADS_PER_BLOCK;
  grid=(n*n%block==0)?n*n/block:(n*n/block+1);

  hipMalloc(&sum_temp, grid*sizeof(double));

  hipEventRecord(start);  
  
  while(count<10){
    // Run kernel on the GPU
    iteration<<<grid,block>>>(d_A,d_B,n);
    hipDeviceSynchronize();
    double *temp;
    temp=d_B;
    d_B=d_A;
    d_A=temp;   
    count++;
    std::cout<<"The numbers of iterations is "<<count<<std::endl;
  }

  size=n*n;
  sum_ptr=d_A;

  while(grid!=0){
    sumblock<<<grid,block,block*sizeof(double)>>>(sum_ptr,size,sum_temp);
    hipDeviceSynchronize();
    size=grid;
    if(grid==1){
      grid=0;
    }
    else{
      grid=(grid%block==0)?grid/block:grid/block+1;
    }    
    sum_ptr=sum_temp;
  }
  
  hipEventRecord(end);
  
  
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsedTime, start, end);

  hipMemcpy(&sum, sum_temp, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&center, (d_A+n/2*n+n/2), sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&verification, (d_A+37*n+47), sizeof(double), hipMemcpyDeviceToHost);


  std::cout <<"The elapsed_time of the cuda program is  "<< elapsedTime << std::endl;
  std::cout <<"The sum of the matrix A after 10 iterations is  "<< sum << std::endl;
  std::cout <<"The center of the matrix A after 10 iterations is  "<< center << std::endl;
  std::cout <<"The A(17,31) of the matrix A after 10 iterations is  "<< verification << std::endl;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(sum_temp);

  free(A);

  return 0;
}
